#include "hip/hip_runtime.h"
/*
 * (c) Crown copyright 2009-2015.
 *
 * This software is distributed under the MIT License.
 * (See accompanying file LICENSE.txt or copy at
 *  http://opensource.org/licenses/MIT)
 */
/*
 * cuda_accel.cpp
 *
 *  Created on: Jan 19, 2010
 *      Author: gareth
 *
 *  Cuda kernels and associated functions
 *
 *  ! must be compiled with nvcc
 *  NB atomic...() functions require -arch=sm_11 flag to nvcc. This is not documented.
 */

#include <hip/hip_runtime.h>
#include "fand/Assert.h"
#include "fand/util.h"
#include "cuda_accel.h"
#include <iostream>
#include <math.h>

#include "fand/MessageStream.h"
INIT_MESSAGES("cuda_accel");
#include "fand/messages.h"

__constant__ CudaLocusInfo loc_info;

#define COPY_OFFSETS() Assert(hipMemcpyToSymbol(HIP_SYMBOL("loc_info"), &locus_info, sizeof(CudaLocusInfo), 0, hipMemcpyHostToDevice) == hipSuccess)

// delta function
#define d(a, b) ((int)(a==b))

__host__
void copyOffsets(CudaLocusInfo const &locus_info)
{
	COPY_OFFSETS();
}

__host__
bool isContext() // is there a CUDA context current in ths thread?
{
//	std::cout << "isContext" << std::endl;
	hipError_t cur;
	hipCtx_t ctx;
	cur = hipCtxPopCurrent(&ctx);
    return (cur == hipSuccess);
}

__host__
void setDevice(int device)
{
//	std::cout << "setDevice" << std::endl;

	// check if there is a CUDA context already
    Assert2( ! isContext(), "hipSetDevice: there is already a context");

    hipError_t err;
    err = hipSetDevice(device);
    Assert2(err == hipSuccess, "hipSetDevice failed");
}

__host__
int getDevice()
{
	int device = -1;
    hipError_t err;
    err = hipGetDevice(&device);
    Assert2(err == hipSuccess, "hipGetDevice failed");
    return device;
}

// triangular number
__host__ __device__
int tri(int i)
{
    return i * (i+1) / 2;
}

// inverse triangle function.
// lower_tri: return greatest i such that tri(i) <= k
// NB 0 < k < tri(N-1)

// implementation 1: binary chop
// NB if we don't know N we use k+1 as our first high value
__host__ __device__
int lower_tri1(int k, int n)
{
    int i_low = 0;
    int i_high = n? (n-1) : (k+1);
    int i;
    while ((i_high - i_low) > 1)
    {
        i = (i_low + i_high)/2;
        if (tri(i) > k)
        {
            i_high = i;
        } else {
            i_low = i;
        }
    }
    return i_low;
}

// implementation 2: quadratic (solve n(n+1) = 2k for n)
// (this looks simpler but is actually slower)
__host__ __device__
int lower_tri2(int k)
{
    float n = (sqrtf(1.0 + 8*k) - 1)/2.0;
    return (int)n;
}

// map from k (the linear index in an upper-triangular DProfile) to i, j coordinates in an n*n matrix
// NB this is NOT the inverse of pindex - but uses a different numbering!
// choose implementation
#define lower_tri lower_tri1
__device__
void getij(int n, int k, int &i, int &j)
{
    i = lower_tri(k, n) + 1;
    j = k - tri(i-1);
}

// map from i, j coordinates to the index in a DProfile
// ( j is along the top: in the upper triangle i <= j)
// Illustrated for n=5:
//  j-->
//     0  1  2  3  4
// i 0 0  1  2  3  4
// | 1    5  6  7  8
// \/2       9 10 11
//   3         12 13
//   5            14
//
// NB this is the same as the order in which elements in an HMatrix h are stored, where
// Allele i = h.m_pmf.first.first
// Allele j = h.m_pmf.first.second

__host__ __device__
int pindex(int i, int j, int n)
{
    return  j + n*i - tri(i);
}

// the folded (upper triangular) index
__host__ __device__
int pindex_ut(int i, int j, int n)
{
    if (i <= j)
    {
        return pindex(i, j, n);
    }
    else
    {
        return pindex(j, i, n);
    }
}

__device__
void storeNResult(NResults *results, int i, float lr)
{
	// first atomically increment the results index
	int n = atomicAdd(&(results->count), 1); // returns old value

	if (n < nresult_max)
	{
		// store results
		results->index[n] = i;
		results->lr[n] = lr;
	}
}

__device__
void storeN2Result(N2Results *results, int i1, int i2, float lr)
{
	// first atomically increment the results index
	int n = atomicAdd(&(results->count), 1);  // returns old value

	if (n < n2result_max)
	{
		// store results
		results->index1[n] = i1;
		results->index2[n] = i2;
		results->lr[n] = lr;
	}
}

// compute element-wise locus1 * locus2 / background and sum
__global__
void cuda_match1(float *loc1, float *loc2, float *back, int n, float *result)
{
// will work in emulator only!
//  printf("blockIdx.x = %d blockDim.x = %d threadIdx.x = %d\n",
//          blockIdx.x, blockDim.x, threadIdx.x);

  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx<n)
  {
	  // to sum this on the device we need to do a reduction (see SDK reduction example)
	  result[idx] = loc1[idx] * loc2[idx] / back[idx];
  }
}

// Calculate LR for two profiles
// Assume all loci are present in array.
// Values <1 signify locus is not present in the array and should be ignored
__device__
float pp_match(DProfile prof1, DProfile prof2, DProfile back, DProfile spmc1 = DProfile())
{
	float profile_lr = 1;

	for (int i=0; i< cuda_num_loci; ++i)
	{
		int locus_size = loc_info.locus_size[i];

		if (locus_size == 0)
		{
			continue; // locus not present in the population database. Skip it.
		}

		int offset = loc_info.locus_offset[i];
		int n_alleles = loc_info.num_alleles[i];

		if ( (prof1.data[offset] < 0) || (prof2.data[offset] < 0) || (back.data[offset] < 0))
		{
			// locus not present in one or both profiles, or the population database. Skip it.
		}
		else
		{

			float locus_lr = 0;

			for (int j=0; j<locus_size; ++j)
			{
				if (back.data[offset + j] > 0)
				{
					locus_lr += prof1.data[offset + j] * prof2.data[offset + j] / back.data[offset + j];
				}
			}

			// Subpopulation correction
			if (spmc1.data != 0)
			{

				float f = 0; // correction factor

				int  p = 0, q = 0; // indices into Dprofiles
				for (int j=0; j<locus_size; ++j)
				{
					// Because we have half-matrices we must divide the off-diagonal elements by 2
					f += spmc1.data[offset + j] * prof2.data[offset + j] / (2 - d(p,q));

					if (q == n_alleles-1)
					{
						++p; q = p;
					}
					else
					{
						++q;
					}
				}

				if (f>0) locus_lr /= f;
			}

			profile_lr *= locus_lr;
		}
	}

	return profile_lr;
}

// this kernel just does one profile/profile match
__global__ void cuda_match2(DProfile prof1, DProfile prof2, DProfile back, float *result)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx>0) return;

	*result = pp_match(prof1, prof2, back);
}

// This kernel compares prof1 with one of the profiles in prof_db
__global__
void cuda_match3(
	DProfile *prof_db,        // profile database
	int n,                    // size of profile database
	DProfile prof1,           // test profile
	DProfile back,            // background
	NResults *results,
	float lr_threshold)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<n)
    {
    	// compare profile idx with prof1

    	DProfile prof2 = prof_db[idx];

		float profile_lr = pp_match(prof1, prof2, back);

		// NB fixed size results table
		// A better solution here is a hash table. We still need atomic writes.
		if (profile_lr > lr_threshold)
		{
			storeNResult(results, idx, profile_lr);
		}
    }
}

// This kernel compares prof1 with ALL of the profiles in prof_db
__global__ void cuda_match4(DProfile *prof_db,        // profile database
		                    int n,                    // size of profile database
		                    DProfile back,            // background
		                    N2Results *results,
		                    float lr_threshold)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<n)
    {
    	// compare profile idx with all other profiles in database

    	DProfile prof2 = prof_db[idx]; // copy into local memory?

    	for (int j=idx+1; j<n; ++j)    // some threads do lots more work than others!
    	{
        	DProfile prof1 = prof_db[j];

        	float profile_lr = pp_match(prof1, prof2, back);

			// NB fixed size results table
			// A better solution here is a hash table. We still need atomic writes.
			if (profile_lr > lr_threshold) // or suitable threshold
			{
				storeN2Result(results, idx, j, profile_lr);
			}
    	}
    }
}

// This kernel can be called in full or half addressing (in full mode half the threads do nothing)
//#define CUDA_MATCH5_HALF

// This kernel compares ONE of the profiles in prof_db with ONE other
__global__ void cuda_match5(DProfile *prof_db,        // profile database
		                    int n,                    // size of profile database
		                    DProfile back,            // background
		                    N2Results *results,
		                    float lr_threshold)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y*blockDim.y + threadIdx.y;

	int i = idx;
	int j = idy;

#ifdef CUDA_MATCH5_HALF
	// In "full-size" mode the kernel is called for an array of size n*n.
	// However only n(n-1)/2 matches are needed, so over half the threads do nothing.

	// In "half-size" mode the kernel is called for an array of size (n/2) * (n-1).
	// Then it is necessary to translate the coordinates of this array into the upper-triangular part of the actual n*n array

	// in fact this is slower! - probably because entire warps do nothing and finish quickly, so waste less
	// time than calling getij

	int k = (n-1) * idx + idy;
	getij(n, k, i, j);
#endif

#ifdef __DEVICE_EMULATION__

	  printf("cuda_match5 kernel: nprofiles = %d\n", n);

	  printf("blockIdx.x = %d blockDim.x = %d threadIdx.x = %d\n",
	          blockIdx.x, blockDim.x, threadIdx.x);

	  printf("blockIdx.y = %d blockDim.y = %d threadIdx.y = %d\n",
	          blockIdx.y, blockDim.y, threadIdx.y);

	  printf("i = %d, j = %d\n", i, j);
#endif

    if (i<n && j<i) // in full-size mode half the threads not used!
    {
    	// re-order to "diagonal traversal" to reduce contention for global memory?
    	// In fact this slows us down!
//    	j = i - (j + 1);

    	// compare profile i with j

		DProfile prof1 = prof_db[j];
    	DProfile prof2 = prof_db[i];

		float profile_lr = pp_match(prof1, prof2, back);

		// NB fixed size results table
		// A better solution here is a hash table. We still need atomic writes.
		if (profile_lr > lr_threshold) // or suitable threshold
		{
			storeN2Result(results, i, j, profile_lr);
		}
    }
}

// This kernel compares ONE of the profiles in prof_db1 with ONE in prof_db2
// This may the same profile, or a derivative of it (such as the corresponding sibling profiles)
// in which case we should perform only the upper-triangular comparison (and n1 == n2)
// Or it may be a different data set, in which case we must do the whole matrix
__global__ void cuda_match6(DProfile *prof_db1,       // profile dataset 1
		                    int n1,                   // size of dataset 1
		                    DProfile *prof_db2,       // profile dataset 2
		                    int n2,                   // size of dataset 2
		                    DProfile back,            // background
		                    ArrayPart part,           //  0 = whole array
		                                              //  1 = upper (excluding diagonal)
		                                              // -1 = lower (including diagonal)
		                    N2Results *results,
		                    float lr_threshold,
		                    int i_start = 0,
		                    int j_start = 0,
		            		DProfile *spmc1_dp = 0)   // SPM correction matrices (for profdb1)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y*blockDim.y + threadIdx.y;

	int i = idx;
	int j = idy;

#ifdef __DEVICE_EMULATION__

	  printf("cuda_match6 kernel: n1 = %d n2 = %d\n", n1, n2);

	  printf("blockIdx.x = %d blockDim.x = %d threadIdx.x = %d\n",
	          blockIdx.x, blockDim.x, threadIdx.x);

	  printf("blockIdx.y = %d blockDim.y = %d threadIdx.y = %d\n",
	          blockIdx.y, blockDim.y, threadIdx.y);
#endif

	// NB in upper/lower mode half the threads not used
   	if ( !(i<n1 && j<n2)                 // not in array
         || (part == upper && !(i_start + i < j_start + j))    // not in upper triangle
         || (part == lower &&  (i_start + i < j_start + j)) )  // not in lower triangle
    {
    	return;
    }

    // compare profile i with j

	DProfile prof1 = prof_db1[i];
	DProfile prof2 = prof_db2[j];
	DProfile spmc1 = spmc1_dp ? spmc1_dp[i] : DProfile();
	float profile_lr = pp_match(prof1, prof2, back, spmc1);

	// NB fixed size results table
	// A better solution here is a hash table. We still need atomic writes.
	if (profile_lr > lr_threshold)
	{
		storeN2Result(results, i_start + i, j_start + j, profile_lr);
	}
}

__host__
void
runCudaMatch6(
		int 			nBlocks1,
		int 			nBlocks2,
		int 			blockSize,
		hipStream_t	stream,
		DProfile 	   *profdb1_d,
		int 			db1_chunk_size,
		DProfile 	   *profdb2_d,
		int 			db2_chunk_size,
		DProfile 		back,
		ArrayPart 		part,
		N2Results	   *results_d,
		float 			lr_threshold,
		int 			i,
		int 			j,
		DProfile 	   *spmc1_dp)
{
	cuda_match6 <<< dim3(nBlocks1, nBlocks2), dim3(blockSize, blockSize), 0, stream >>>
			(profdb1_d, db1_chunk_size, profdb2_d, db2_chunk_size, back, part, results_d, lr_threshold, i, j, spmc1_dp);
}

// like cuda_match6 but tiled. One locus at a time to allow shared memory.
//
// 2D Grid 1D block
//
//#define SHARED
__global__
void cuda_match7(
	DProfile *prof_db1,       // profile dataset 1
	int n1,                   // size of dataset 1
	DProfile *prof_db2,       // profile dataset 2
	int n2,                   // size of dataset 2
	DProfile back,            // background
	bool upper,
	N2Results *results,
	float lr_threshold)
{
	int i = threadIdx.x;
	int idx = blockIdx.x*blockDim.x + i;

#ifdef __DEVICE_EMULATION__

	  printf("\ncuda_match7 kernel: n1 = %d n2 = %d\n", n1, n2);

	  printf("blockIdx.x = %d blockDim.x = %d threadIdx.x = %d\n",
	          blockIdx.x, blockDim.x, threadIdx.x);

	  printf("blockIdx.y = %d blockDim.y = %d threadIdx.y = %d\n",
	          blockIdx.y, blockDim.y, threadIdx.y);

	  printf("i = %d idx = %d\n", i, idx);
#endif


	// do one locus at a time

	// shared memory copies of the current locus
#ifdef SHARED
	__shared__ float locus_p1[cuda_locus_size * blockSize];
	__shared__ float locus_p2[cuda_locus_size * blockSize];
	__shared__ float locus_back[cuda_locus_size * blockSize];
#endif

	// results
	__shared__ float lr[blockSize][blockSize];
	for (int j=0; j<blockSize; ++j)
	{
		lr[i][j] = 1;
	}

	for (int loc=0; loc<cuda_num_loci; ++loc)
	{
		int offset = loc_info.locus_offset[loc];
		int locus_size = loc_info.locus_size[loc];

		if (locus_size == 0)
		{
			// TODO: locus not present in population database. What to do?
		}

#ifdef __DEVICE_EMULATION__
		  printf("locus = %d offset = %d\n", loc, offset);
#endif
		//
		// copy locus loc of each profile into shared memory
		//
#ifdef SHARED
		// This memcpy is very slow! is there a faster way to do it?
#if 0
		memcpy(locus_p1 + cuda_locus_size * i, prof_db1[blockIdx.x*blockDim.x + i].data + offset, cuda_locus_size * sizeof(float));
		memcpy(locus_p2 + cuda_locus_size * i, prof_db2[blockIdx.y*blockDim.x + i].data + offset, cuda_locus_size * sizeof(float));
		memcpy(locus_back + cuda_locus_size * i, back.data + offset, cuda_locus_size * sizeof(float));
#else
		float *d1 = locus_p1 + cuda_locus_size * i;
		float *d2 = locus_p2 + cuda_locus_size * i;
		float *d3 = locus_back + cuda_locus_size * i;
		float *s1 = prof_db1[blockIdx.x*blockDim.x + i].data + offset;
		float *s2 = prof_db2[blockIdx.y*blockDim.x + i].data + offset;
		float *s3 = back.data + offset;

		int c = cuda_locus_size;
		while (c--)
		{
			*d1++ = *s1++;
			*d2++ = *s2++;
			*d3++ = *s3++;
		}
#endif

#endif
		// when everyone has caught up all the data for this locus, in this grid element will have been copied
		__syncthreads();

		//
		// do a row of matches
		//
		for (int j=0; j<blockSize; ++j)
		{
			int idy = blockIdx.y*blockDim.x + j; // NB blockDim.x not blockDim.y

#ifdef __DEVICE_EMULATION__
			printf("j = %d idy = %d\n", j, idy);
#endif

			if (upper && ((n1!=n2) || !(idy<n1 && idx<idy))) // in upper mode half the threads not used
			{
				lr[i][j] = 0;
#ifdef __DEVICE_EMULATION__
			printf("nowt to do\n");
#endif
				continue;
			}

#ifndef SHARED
			//
			// Compare loci in global memory
			//
			DProfile prof1 = prof_db1[idx];
			DProfile prof2 = prof_db2[idy];

			if ( (prof1.data[offset] < 0) || (prof2.data[offset] < 0) )
			{
				// locus not present in one or both profiles
	//			continue;
			}
			else
			{

				float locus_lr = 0;

				for (int k=0; k<locus_size; ++k)
				{
					if (back.data[offset + k] > 0)
					{
						locus_lr += prof1.data[offset + k] * prof2.data[offset + k] / back.data[offset + k];
					}
				}

				lr[i][j] *= locus_lr;
			}
#else
			//
			// Compare loci in shared memory
			//
			float *loc1 = locus_p1 + cuda_locus_size * i;
			float *loc2 = locus_p2 + cuda_locus_size * j;


			if ( (*loc1 < 0) || (*loc2 < 0) )
			{
				// locus not present in one or both profiles
			}
			else
			{
				float locus_lr = 0;

				for (int k=0; k<cuda_locus_size; ++k)
				{
					if (locus_back[k] > 0)
					{
						locus_lr += loc1[k] * loc2[k] / locus_back[k];
					}
				}

				lr[i][j] *= locus_lr;
			}
#endif
		}
	}

	// row complete - report results
	for (int j=0; j<blockSize; ++j)
	{
		int idy = blockIdx.y*blockDim.x + j; // NB blockDim.x not blockDim.y

#ifdef __DEVICE_EMULATION__
	  printf("lr[%d][%d] = %f ", i, j, lr[i][j]);
#endif
		if (lr[i][j] > lr_threshold)
		{
			storeN2Result(results, idx, idy, lr[i][j]);
		}
	}
#ifdef __DEVICE_EMULATION__
	  printf("\n");
#endif
}

// construct the contribution of prof to an n-degree descendant
// (equivalently, the contribution from an n-degree ancestor)
// This is the PMF of the allele contributed by prof.
// NB the other allele will be 'F' (background) unless this
// is a two-path descendant. n == 0 represents infinite dilution,
// and returns 'F'
//
// This amounts to evaluating (1/2^n) A + (1/2^n) B + (1 - 2/(2^n)) F
//
__host__ __device__
void allele_pmf(
	int n,                      // degree of relationship
    int i,                      // allele index
    int j,                      // allele index
    int size,                   // size of PMF
    const float back[],         // background
    float pmf[])                // allele PMF (to construct)
{
	float wa = (n==CUDAMatchType::INF) ? 0 : 1.0/pow(2.0, n); // weight of alleles
	float wf = 1 - 2*wa;                                      // weight of background

	for (int k=0; k<size; ++k)
	{
		pmf[k] = back[k] * wf;
	}

	pmf[i] += wa;
	pmf[j] += wa;
}

// construct the vector a * A + b * B + (1-(a+b)) F
__host__ __device__
void gen_allele_pmf(
	float wi,                   // proportion of 'Ai'
	float wj,                   // proportion of 'Aj'
    int i,                      // allele index
    int j,                      // allele index
    int size,                   // size of PMF
    const float back[],         // background
    float pmf[])                // allele PMF (to construct)
{
	float wf = 1 - wi - wj;     // weight of background

	for (int k=0; k<size; ++k)
	{
		pmf[k] = back[k] * wf;
	}

	pmf[i] += wi;
	pmf[j] += wj;
}

__host__ __device__
void
make_sib(
	DProfile prof,       // profile dataset
    DBackground back,    // background vector
	DProfile psib,       // sibling dataset (to construct)
	CudaLocusInfo const *loc_info)
{
    //
    // construct sib_db from prof_db. Method is as in match.cpp: sib()
    //
	for (int loc=0; loc< cuda_num_loci; ++loc)
	{
		int locus_size = loc_info->locus_size[loc];

		if (locus_size == 0) // locus not in population database
		{
			continue;
		}

		int loc_offset = loc_info->locus_offset[loc];
		int back_offset = loc_info->back_offset[loc];

		// if the input profile is -1 (locus not present), then so is the sibling
		if (prof.data[loc_offset] < 0)
		{
			for (int j=0; j<locus_size; ++j)
			{
				psib.data[loc_offset + j] = -1;
			}
			continue;
		}

		int n_alleles = loc_info->num_alleles[loc];

		float Pq[CUDA_MAX_ALLELES]; // Sum_q Pqj
		float Pr[CUDA_MAX_ALLELES]; // Sum_r Pir

		for (int i=0; i<n_alleles; ++i)
		{
			Pq[i] = 0; Pr[i] = 0;
		}

		// Sum rows and columns and zero the result
		// (Here we need only consider non-zero elements of P,
		// i.e. in the upper triangle).
		int k = 0; // index into prof
		for (int i=0; i<n_alleles; ++i)
		{
			for (int j=i; j<n_alleles; ++j)
			{
				float p = prof.data[loc_offset + k];

				Pq[j] += p;
				Pr[i] += p;

				psib.data[loc_offset + k] = 0;
				++k;
			}
		}

		// This needs to be a sum over all (i, j).
		// But remember prof and sib are upper-triangular
		for (int j=0; j<n_alleles; ++j)
		{
			for (int i=0; i<n_alleles; ++i)
			{
                int k = pindex_ut(i, j, n_alleles); // the 'folded' index into prof/sib

				float Sij = 0;

				// b(i)b(j)
				Sij += back.data[back_offset + i] * back.data[back_offset + j];

				// b(i) * Sum_q Pqj
				Sij += back.data[back_offset + i] * Pq[j];

				// b(j) * Sum_r Pir
				Sij += back.data[back_offset + j] * Pr[i];

				// Pij. NB since prof is a half-matrix we must take the value as 0 below the diagonal,
				// or we will double-count
				if (i <= j)
				{
					Sij += prof.data[loc_offset + k];
				}

				Sij *= 0.25;

				psib.data[loc_offset + k] += Sij;
			}
		}
	}
}

// Beta-binomial sampling formula:
// Probability of Allele A given that y out of n alleles observed were A; HW freq P(A) = f
__host__ __device__
float bbsf(int y,
           int n,
           float fst,
           float f)
{
    float ret = (y * fst + (1-fst) * f) / (1 + (n-1) * fst);
    return ret;
}

// probability a new allele is p given i, j observed
__host__ __device__
float
prob_BNp_giv_ij(
		int p,
		int i,
		int j,
        float fp,
        float fst)
{
    // P(p|ij)

    int y = d(p,i) + d(p,j);
    return bbsf(y, 2, fst, fp);
}

// probability p and a new allele is i, j given p, q observed
__host__ __device__
float
prob_BNpX_is_ij(
	int p,
	int q,
	int i,
	int j,
    float fi,
    float fj,
    float fst)
{
    // P(pX == ij) | pq where X is drawn from the background

    if (p==i)
    {
        return prob_BNp_giv_ij(j,p,q,fj,fst); // P(j|pq)
    }
    else if (p==j)
    {
        return prob_BNp_giv_ij(i,p,q,fi,fst); // P(i|pq)
    }
    else
    {
        return 0;
    }
}

// probability two new alleles are p,q given i,j observed
// (NB this is a generalized version of NRC4_10)
__host__ __device__
float
prob_BNpq_giv_ij(
		int p,
		int q,
		int i,
		int j,
        float fp,
        float fq,
        float fst)
{
    // P(pq|ij) = P(p|ij) * P(q|pij) : *2 if p!=q

    int y1 = d(p,i) + d(p,j);
    int y2 = d(q,p) + d(q,i) + d(q,j);

    float x = bbsf(y1, 2, fst, fp) * bbsf(y2, 3, fst, fq);

    if (p != q) x *= 2;

    return x;
}

// P(Rel(pq) = ij)
__host__ __device__
float
rel_BN(
      int p,
      int q,
      int i,
      int j,
      float fi,
      float fj,
      float k0,    // no alleles Identical By Descent
      float k1,    // one allele IBD
      float k2d,   // two alleles (different) IBD
//    float k2s,   // two alleles (the same) IBD // TODO - needed for bilineal
      float fst)
{
    return    k2d   * d(p,i) * d(q,j)                        // P(pq == ij)
           + (k1/2) * (prob_BNpX_is_ij(p,q,i,j,fi,fj,fst) +  // P(p? == ij)
                       prob_BNpX_is_ij(q,p,i,j,fi,fj,fst) )  // P(q? == ij)
           +  k0    *  prob_BNpq_giv_ij(i,j,p,q,fi,fj,fst);  // P(?? == ij) == P(ij|pq)
}

__host__ __device__
void
make_rel_BN(
	DProfile prof,       // profile dataset
    DBackground back,    // background vector
	DProfile psib,       // sibling dataset (to construct)
	CudaLocusInfo const *loc_info,
	float k0,              // no alleles Identical By Descent
	float k1,              // one allele IBD
	float k2d,             // two alleles (different) IBD
//      float k2s,         // two alleles (the same) IBD // TODO - needed for bilineal
	float theta)
{
    //
    // construct sib_db from prof_db.
	//
    // For all (loc):
    //   For all (ij):
	//     psib(ij) = 0
    //
    //   For all (pq): // elements of prof
	//     w = prof(pq)
	//     if (w>0)
    // 	     For all (ij): // elements of psib
	//         fi = back(i)
	//         fj = back(j)
    //         psib(ij) += w * P(Rel(pq) = ij)
	//
	for (int loc=0; loc< cuda_num_loci; ++loc)
	{
		int locus_size = loc_info->locus_size[loc];

		if (locus_size == 0) // locus not in population database
		{
			continue; // next locus
		}

		int loc_offset  = loc_info->locus_offset[loc];
		int back_offset = loc_info->back_offset[loc];
		int n_alleles   = loc_info->num_alleles[loc];

		// if the input profile is -1 (locus not present), then so is the sibling
		if (prof.data[loc_offset] < 0)
		{
			for (int k=0; k<locus_size; ++k)
			{
				psib.data[loc_offset + k] = -1;
			}
			continue; // next locus
		}

		// zero psib
		for (int k=0; k<locus_size; ++k)
		{
			psib.data[loc_offset + k] = 0;
		}

		// loop over elements of prof (which is upper-triangular)
		// *in the order of the DProfile index*
		int n = 0; // index into prof
		for (int p=0; p<n_alleles; ++p)
		{
			for (int q=p; q<n_alleles; ++q)
			{
				float w = prof.data[loc_offset + n]; // weight of this element of prof

				if (w > 0)
				{
					// loop over elements of psib *in the order of the DProfile index*
					int k = 0; // index into psib
					for (int i=0; i<n_alleles; ++i)
					{
						for (int j=i; j<n_alleles; ++j)
						{
							float fi = back.data[back_offset + i];
							float fj = back.data[back_offset + j];

							// P( Rel(pq) = ij )
							float x = rel_BN(p, q, i, j, fi, fj, k0, k1, k2d, theta);
							psib.data[loc_offset + k] += w * x;
							++k;
						}
					}
				}
				++n;
			}
		}
	}
}

// construct the (n, m) two-path relative of prof
// Use m=MatchType::INF for a one-path relationship

__host__ __device__
void
make_2path(
	DProfile prof,       // profile dataset
	int n,               // first degree of relationship
	int m,               // second degree of relationship (MatchType::INF == F)
    DBackground back,    // background vector
	DProfile prel,       // relative dataset (to construct)
	CudaLocusInfo const *loc_info)
{
	// At each locus, for each element of prof, construct a degree n and a degree m and multiply them
	// TODO this is the 'long hand' method - on CUDA we should reduce the sum algebraically as we do for sibling and this will be faster
	// We can use this implementation for comparison.

	for (int loc=0; loc< cuda_num_loci; ++loc)
	{
		int loc_offset = loc_info->locus_offset[loc];
		int back_offset = loc_info->back_offset[loc];
		int locus_size = loc_info->locus_size[loc];

		if (locus_size == 0) // locus not in population database
		{
			continue;
		}

		// if the input profile is -1 (locus not present), then so is the relative
		if (prof.data[loc_offset] < 0)
		{
			for (int j=0; j<locus_size; ++j)
			{
				prel.data[loc_offset + j] = -1;
			}
			continue;
		}

		int n_alleles = loc_info->num_alleles[loc];

		float Pn[CUDA_MAX_ALLELES]; // n-degree distribution
		float Pm[CUDA_MAX_ALLELES]; // m-degree distribution

		// loop over elements of prof (which is upper-triangular)
		for (int ip=0; ip<n_alleles; ++ip)
		{
			for (int jp=ip; jp<n_alleles; ++jp)
			{
				int p = pindex(ip, jp, n_alleles);
				float wp = prof.data[loc_offset + p]; // weight of this element of prof

				if (wp > 0)
				{
					// construct n-degree and m-degree allele distributions for this element
					allele_pmf(n, ip, jp, n_alleles, back.data + back_offset, Pn);
					allele_pmf(m, ip, jp, n_alleles, back.data + back_offset, Pm);

					// multiply and add in to the result
					for (int j=0; j<n_alleles; ++j)
					{
						for (int i=0; i<n_alleles; ++i)
						{
			                int k = pindex_ut(i, j, n_alleles); // the 'folded' index into prof/prel

							prel.data[loc_offset + k] += Pn[i] * Pm[j] * wp;
						}
					}
				}
			}
		}
	}
}

__host__ __device__
void
make_genrc_elem(
	float wp,            // element weight
	int i,               // element i
	int j,               // element j (>=i)
	int n_alleles,       // alleles at this locus
	float a1,            // proportion of 'a' inherited on side 1
	float b1,            // proportion of 'b' inherited on side 1
	float a2,            // proportion of 'a' inherited on side 2
	float b2,            // proportion of 'b' inherited on side 2
    float *pback,        // background element
    float *prel)         // relative element (to construct)
{
	float P1[CUDA_MAX_ALLELES];   // vector for side 1, as given
	float P2[CUDA_MAX_ALLELES];   // vector for side 2, as given
	float P1_s[CUDA_MAX_ALLELES]; // vector for side 1, a's and b's swapped
	float P2_s[CUDA_MAX_ALLELES]; // vector for side 2, a's and b's swapped

	if (wp > 0)
	{
		wp /= 2; // because we must average two components to form the matrix

		// construct vectors as given
		gen_allele_pmf(a1, b1, i, j, n_alleles, pback, P1);
		gen_allele_pmf(a2, b2, i, j, n_alleles, pback, P2);

		// construct vectors with a's and b's swapped
		gen_allele_pmf(b1, a1, i, j, n_alleles, pback, P1_s);
		gen_allele_pmf(b2, a2, i, j, n_alleles, pback, P2_s);

		// multiply and add in to the result
		for (int p=0; p<n_alleles; ++p)
		{
			for (int q=0; q<n_alleles; ++q)
			{
				int k = pindex_ut(p, q, n_alleles); // the 'folded' index into prof/prel

				prel[k] += (P1[p] * P2[q] + P1_s[p] * P2_s[q]) * wp;
			}
		}
	}
}

// construct the [(a1, b1), (a2, b2)] generalized relationship coefficient relative of prof
//
// NB since we do not know which parent an allele comes from, we must always
// 1) form a matrix with the numbers as given
// 2) form a matrix with a1/b1 swapped and a2/b2 swapped
// 3) average them
//
// NB the matrices are symmetrical (upper triangular) which takes care of the side-1/side-2 symmetry
//
__host__ __device__
void
make_genrc(
	DProfile prof,       // profile dataset
	float a1,            // proportion of 'a' inherited on side 1
	float b1,            // proportion of 'b' inherited on side 1
	float a2,            // proportion of 'a' inherited on side 2
	float b2,            // proportion of 'b' inherited on side 2
    DBackground back,    // background vector
	DProfile prel,       // relative dataset (to construct)
	CudaLocusInfo const *loc_info)
{
	for (int loc=0; loc< cuda_num_loci; ++loc)
	{
		int loc_offset = loc_info->locus_offset[loc];
		int back_offset = loc_info->back_offset[loc];
		int locus_size = loc_info->locus_size[loc];

		if (locus_size == 0) // locus not in population database
		{
			continue;
		}

		// if the input profile is -1 (locus not present), then so is the relative
		if (prof.data[loc_offset] < 0)
		{
			for (int j=0; j<locus_size; ++j)
			{
				prel.data[loc_offset + j] = -1;
			}
			continue;
		}

		int n_alleles = loc_info->num_alleles[loc];

		// loop over elements of prof (which is upper-triangular)
		for (int ip=0; ip<n_alleles; ++ip)
		{
			for (int jp=ip; jp<n_alleles; ++jp)
			{
				int p = pindex(ip, jp, n_alleles);
				float wp = prof.data[loc_offset + p]; // weight of this element of prof

				make_genrc_elem(wp, ip, jp, n_alleles, a1, b1, a2, b2, back.data + back_offset, prel.data + loc_offset);
			}
		}
	}
}

//
// make the inverse of the given 4-number relationship
// (Bayes theorem - see maths paper)
//
__host__ __device__
void
make_invrc(
	DProfile prof,       // profile dataset
	float a1,            // proportion of 'a' inherited on side 1
	float b1,            // proportion of 'b' inherited on side 1
	float a2,            // proportion of 'a' inherited on side 2
	float b2,            // proportion of 'b' inherited on side 2
    DBackground back,    // background vector
	DProfile prel,       // relative dataset (to construct)
	CudaLocusInfo const *loc_info)
{
	for (int loc=0; loc< cuda_num_loci; ++loc)
	{
		int loc_offset = loc_info->locus_offset[loc];
		int back_offset = loc_info->back_offset[loc];
		int locus_size = loc_info->locus_size[loc];

		if (locus_size == 0) // locus not in population database
		{
			continue;
		}

		// if the input profile is -1 (locus not present), then so is the relative
		if (prof.data[loc_offset] < 0)
		{
			for (int j=0; j<locus_size; ++j)
			{
				prel.data[loc_offset + j] = -1;
			}
			continue;
		}

		int n_alleles = loc_info->num_alleles[loc];

		for (int i=0; i<n_alleles; ++i)
		{
			for (int j=i; j<n_alleles; ++j)
			{
				const int size = CUDA_MAX_ALLELES * (CUDA_MAX_ALLELES + 1) / 2; // auto arrays must be const size
				float R[size] = { 0 }; // should initialize whole array to zero ...

				// ... but doesn't, so
				int actual_size = n_alleles * (n_alleles + 1) / 2;
				for (int k=0; k<actual_size; ++k)
				{
				    R[k] = 0;
				}

				int k_ij = pindex(i, j, n_alleles);

				float b_ij = (back.data + back_offset)[i] * (back.data + back_offset)[j]
														  * (i==j ? 1 : 2);

				const float AiAj = 1.0;
				make_genrc_elem(AiAj, i, j, n_alleles, a1, b1, a2, b2, back.data + back_offset, R);

				float prel_elem = 0;

				for (int p=0; p<n_alleles; ++p)
				{
					for (int q=p; q<n_alleles; ++q)
					{
						int k_pq = pindex(p, q, n_alleles);

						float b_pq = (back.data + back_offset)[p] * (back.data + back_offset)[q]
																  * (p==q ? 1 : 2);

						prel_elem += prof.data[loc_offset + k_pq] * R[k_pq] / b_pq;
					}
				}

				prel.data[loc_offset + k_ij] = prel_elem * b_ij;
			}
		}
	}
}

// Construct Generalized Relationship Coefficient relative profiles (non-sibling)
__global__ void cuda_genrc(DProfile *prof_db,      // profile dataset
						  DProfile *rel_db,        // relative dataset (to construct)
		                  int n,                   // size of dataset
 		             	  float a1,                // proportion of 'a' inherited on side 1
		             	  float b1,            	   // proportion of 'b' inherited on side 1
		             	  float a2,                // proportion of 'a' inherited on side 2
		             	  float b2,                // proportion of 'b' inherited on side 2
		                  DBackground back,        // background
		                  bool inverse = false)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i>=n)
    {
    	return;
    }

    // initialize rel_db[i] to zero
	for (int j=0; j<prof_db[i].size; ++j)
	{
		rel_db[i].data[j] = 0;
	}

    // construct rel_db[i] from prof_db[i]
	if (inverse)
	{
		make_invrc(prof_db[i], a1, b1, a2, b2, back, rel_db[i], &loc_info);
	}
	else
	{
		make_genrc(prof_db[i], a1, b1, a2, b2, back, rel_db[i], &loc_info);
	}
}

// Construct (d1, d2) relative profiles (non-sibling)
__global__ void cuda_rel(DProfile *prof_db,       // profile dataset
						 DProfile *rel_db,        // relative dataset (to construct)
		                 int n,                   // size of dataset
		                 int d1,                  // degree first path
		                 int d2,                  // degree second path (NOT HANDLED)
		                 DBackground back,        // background
		                 CudaSubPopModel spm)
//		                 float theta)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i>=n)
    {
    	return;
    }

    if (spm.type == CudaSubPopModel::HW)
    {
        // initialize rel_db[i] to zero
        // NB memset needs a constant size. We could use cuda_prof_size but watch out if this is no longer constant!
    //	memset(rel_db[i].data, 0, cuda_prof_size * sizeof(float));
    	for (int j=0; j<prof_db[i].size; ++j)
    	{
    		rel_db[i].data[j] = 0;
    	}

        // construct rel_db[i] from prof_db[i].
        make_2path(prof_db[i], d1, d2, back, rel_db[i], &loc_info);

    }
    else // Balding-Nichols (we do not support 4.4 for relatives)
    {
    	// IBD coefficients for 1-path:
    	float k1 = pow(2.0, 1-d1);
    	float k0 = 1 - k1;
    	float k2d = 0;

    	make_rel_BN(prof_db[i], back, rel_db[i], &loc_info, k0, k1, k2d, spm.theta_bar);
    }
}

__device__
void
make_spmc(
	DProfile prof,       // profile dataset
	DProfile back,       // background matrix (4.4)
    DBackground backv,   // background vector (HW)
	DProfile spmc,       // SPM correction matrices (to construct)
	float theta,         //  Fst
	CudaLocusInfo const *loc_info)
{
    // for each locus, construct a correction matrix
	for (int loc=0; loc< cuda_num_loci; ++loc)
	{
		int locus_size = loc_info->locus_size[loc];

		if (locus_size == 0) // locus not in population database
		{
			continue;
		}

		int loc_offset = loc_info->locus_offset[loc];
		int back_offset = loc_info->back_offset[loc];

		// if the input profile is -1 (locus not present), then so is the correction matrix
		if (prof.data[loc_offset] < 0)
		{
			for (int j=0; j<locus_size; ++j)
			{
				spmc.data[loc_offset + j] = -1;
			}
			continue;
		}

		int n_alleles = loc_info->num_alleles[loc];

		// loop over all genotypes
		int k = 0; // index into prof, spmc etc
		for (int i=0; i<n_alleles; ++i)
		{
			for (int j=i; j<n_alleles; ++j)
			{
				float x = 0;
				int m = 0;
				for (int p=0; p<n_alleles; ++p)
				{
					for (int q=p; q<n_alleles; ++q)
					{
				        float fp = backv.data[back_offset + p]; // HW frequency of p
				        float fq = backv.data[back_offset + q]; // HW frequency of q

						float Bpq_ij = prob_BNpq_giv_ij(p, q, i, j, fp, fq, theta);

						x += prof.data[loc_offset + m] * Bpq_ij / back.data[loc_offset + m];
						++m;
					}
				}

				spmc.data[loc_offset + k] = (i==j) ? x : 2*x; // *2 because a half-matrix
				++k;
			}
		}
	}
}

// Construct correction matrices
__global__ void cuda_spmc(DProfile *prof_db,       // profile dataset
						  DProfile *spmc_db,       // SPM correction matrix (to construct)
		                  int n,                   // size of dataset
		              	  DProfile back,           // background matrix (4.4)
		                  DBackground backv,       // background vector (HW)
		                  float theta)             // Fst
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i>=n)
    {
    	return;
    }

    make_spmc(prof_db[i], back, backv, spmc_db[i], theta, &loc_info);
}

// Construct sibling profiles
__global__ void cuda_sib(DProfile *prof_db,       // profile dataset
						 DProfile *sib_db,        // sibling dataset (to construct)
		                 int n,                   // size of dataset
		                 DBackground back,        // background
		                 CudaSubPopModel spm)
//		                 float theta)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i>=n)
    {
    	return;
    }

    // construct sib_db[i] from prof_db[i].
    if (spm.type == CudaSubPopModel::HW)
    {
    	make_sib(prof_db[i], back, sib_db[i], &loc_info);
    }
    else // Balding-Nichols (we do not support 4.4 for relatives)
    {
    	// IBD coefficients for sibling:
    	float k0 = 0.25;
    	float k1 = 0.5;
    	float k2d = 0.25;

    	make_rel_BN(prof_db[i], back, sib_db[i], &loc_info, k0, k1, k2d, spm.theta_bar);
    }

}

#ifndef STREAMED
void cuda_n2match(std::vector<DProfile> &prof_db, // Profile database
				  DProfile back,                  // Background
				  CudaLocusInfo const &locus_info,
				  N2Results &results,
				  float lr_threshold)
{
	int n_floats_per_profile = back.size;
	int n_profiles = prof_db.size();
	int n_floats = n_floats_per_profile * n_profiles;
	Assert(n_floats_per_profile == locus_info.profile_size);

	//
	// allocate DProfiles data and result on device
	//
	Timer t;

	// the result
	N2Results *results_d;
	Assert2(hipMalloc((void **) &results_d, sizeof(N2Results)*1) != hipErrorOutOfMemory,
			"cuda_n2match: A: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2match: B: hipDeviceSynchronize failed");
	Assert(results_d);

	// the data in the profiles
	float *profdb_data = 0;
	Assert2(hipMalloc((void **) &profdb_data, sizeof(float)*n_floats) != hipErrorOutOfMemory,
			"cuda_n2match: C: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2match: D: hipDeviceSynchronize failed");
	Assert(profdb_data);

	// the DProfiles themselves (containing pointers to the data)
	DProfile *profdb_d = 0;
	Assert2(hipMalloc((void **) &profdb_d, sizeof(DProfile)*n_profiles) != hipErrorOutOfMemory,
			"cuda_n2match: E: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2match: F: hipDeviceSynchronize failed");
	Assert(profdb_d);

	// the data for the background
	float *back_data = 0;
	Assert2(hipMalloc((void **) &back_data, sizeof(float)*n_floats_per_profile) != hipErrorOutOfMemory,
			"cuda_n2match: G: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2match: H: hipDeviceSynchronize failed");
	Assert(back_data);

    t.stop();
    info << startl << "cuda_n2match(): Allocating memory on device took " << t << " seconds" << std::endl;

	//
	// copy to device once only
	//
    t.start();

    // copy offsets
    COPY_OFFSETS();

	// zero the result
	results.count = 0;
	hipMemcpy(results_d, &results, sizeof(NResults)*1, hipMemcpyHostToDevice);

	// Data for each DProfile. After copying each data array, copy the device address into the DProfile.
	for (int i=0; i<n_profiles; ++i)
	{
		float *addr = profdb_data + (i * n_floats_per_profile);
		hipMemcpy(addr, prof_db[i].data, sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
		prof_db[i].data = addr;
	}

	// the DProfiles themselves (containing pointers to the data)
	hipMemcpy(profdb_d, &(prof_db[0]), sizeof(DProfile)*n_profiles, hipMemcpyHostToDevice);

	// the data for the background
	hipMemcpy(back_data,   back.data,  sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
	back.data  = back_data;

	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2match: I: hipDeviceSynchronize failed");

    t.stop();
    info << startl << "cuda_n2match(): Copying data to device took " << t << " seconds" << std::endl;

    //
	// Call match kernel
    //
    t.start();

    info << startl << "launching kernel cuda_match6" << std::endl;
    int nBlocks = n_profiles/blockSize + (n_profiles%blockSize == 0?0:1);
    info << alignl << "nBlocks = " << nBlocks << std::endl;
    info << alignl << "blockSize = " << blockSize << std::endl;
    info << alignl << "n_profiles = " << n_profiles << std::endl;

#if 1
    cuda_match6 <<< dim3(nBlocks, nBlocks), dim3(blockSize, blockSize) >>>
    		(profdb_d, n_profiles, profdb_d, n_profiles, back, upper, results_d, lr_threshold);
#else
    cuda_match7 <<< dim3(nBlocks, nBlocks), dim3(blockSize) >>>
    		(profdb_d, n_profiles, profdb_d, n_profiles, back, true, results_d, lr_threshold);
#endif

    hipError_t cts = hipDeviceSynchronize();
    info << startl << "hipDeviceSynchronize() == " << cts << std::endl;
    hipError_t err = hipGetLastError();
    info << startl << "hipGetLastError() == " << err << std::endl;

    Assert2(cts == hipSuccess, "cuda_n2match: J: hipDeviceSynchronize failed");

    t.stop();
    info << startl << "cuda_n2match(): Kernel cuda_match6 took " << t << " seconds" << std::endl;

    // get result from device
    hipMemcpy(&results, results_d, sizeof(N2Results)*1, hipMemcpyDeviceToHost);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2match: K: hipDeviceSynchronize failed");

    info << startl << min(results.count, n2result_max) << " results copied from device" << std::endl;

	// clean up
	hipFree(results_d);
	hipFree(profdb_data);
	hipFree(profdb_d);
	hipFree(back_data);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2match: L: hipDeviceSynchronize failed");
}

// match one db against another (not necessarily the same size)
void cuda_nm_match(std::vector<DProfile> &prof_db1, // Profile database
				   std::vector<DProfile> &prof_db2, // Profile database
				   DProfile back,                   // Background
 	 			   CudaLocusInfo const &locus_info,
				   N2Results &results,
				   float lr_threshold,
				   ArrayPart part)
{
	int n_floats_per_profile = back.size;
	int n1 = prof_db1.size();
	int n2 = prof_db2.size();
	int n_floats1 = n_floats_per_profile * n1;
	int n_floats2 = n_floats_per_profile * n2;
	Assert(n_floats_per_profile == locus_info.profile_size);

//	Assert(part == full || n1 == n2);

	//
	// allocate DProfiles data and result on device
	//
	Timer t;

	// the result
	N2Results *results_d;
	Assert2(hipMalloc((void **) &results_d, sizeof(N2Results)*1) != hipErrorOutOfMemory,
			"cuda_nm_match: AA: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_match: A: hipDeviceSynchronize failed");
	Assert(results_d);

	// the data in the profiles
	float *profdb1_data = 0;
	Assert2(hipMalloc((void **) &profdb1_data, sizeof(float)*n_floats1) != hipErrorOutOfMemory,
			"cuda_nm_match: B: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_match: C: hipDeviceSynchronize failed");
	Assert(profdb1_data);

	float *profdb2_data = 0;
	Assert2(hipMalloc((void **) &profdb2_data, sizeof(float)*n_floats2) != hipErrorOutOfMemory,
			"cuda_nm_match: D: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_match: E: hipDeviceSynchronize failed");
	Assert(profdb2_data);

	// the DProfiles themselves (containing pointers to the data)
	DProfile *profdb1_d = 0;
	Assert2(hipMalloc((void **) &profdb1_d, sizeof(DProfile)*n1) != hipErrorOutOfMemory,
			"cuda_nm_match: F: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_match: G: hipDeviceSynchronize failed");
	Assert(profdb1_d);

	DProfile *profdb2_d = 0;
	Assert2(hipMalloc((void **) &profdb2_d, sizeof(DProfile)*n2) != hipErrorOutOfMemory,
			"cuda_nm_match: H: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_match: I: hipDeviceSynchronize failed");
	Assert(profdb2_d);

	// the data for the background
	float *back_data = 0;
	Assert2(hipMalloc((void **) &back_data, sizeof(float)*n_floats_per_profile) != hipErrorOutOfMemory,
			"cuda_nm_match: J: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_match: K: hipDeviceSynchronize failed");
	Assert(back_data);

    t.stop();
    info << startl << "cuda_nm_match(): Allocating memory on device took " << t << " seconds" << std::endl;

	//
	// copy to device
	//
    t.start();

    // copy offsets
    COPY_OFFSETS();

	// zero the result
	results.count = 0;
	hipMemcpy(results_d, &results, sizeof(NResults)*1, hipMemcpyHostToDevice);

	// Data for each DProfile. After copying each data array, copy the device address into the DProfile.
	for (int i=0; i<n1; ++i)
	{
		float *addr = profdb1_data + (i * n_floats_per_profile);
		hipMemcpy(addr, prof_db1[i].data, sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
		prof_db1[i].data = addr;
	}

	for (int i=0; i<n2; ++i)
	{
		float *addr = profdb2_data + (i * n_floats_per_profile);
		hipMemcpy(addr, prof_db2[i].data, sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
		prof_db2[i].data = addr;
	}

	// the DProfiles themselves (containing pointers to the data)
	hipMemcpy(profdb1_d, &(prof_db1[0]), sizeof(DProfile)*n1, hipMemcpyHostToDevice);
	hipMemcpy(profdb2_d, &(prof_db2[0]), sizeof(DProfile)*n2, hipMemcpyHostToDevice);

	// the data for the background
	hipMemcpy(back_data,   back.data,  sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
	back.data  = back_data;

	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_match: L: hipDeviceSynchronize failed");

    t.stop();
    info << startl << "cuda_nm_match(): Copying data to device took " << t << " seconds" << std::endl;

    //
	// Call solution kernel
    //
    t.start();

    info << startl << "launching kernel cuda_match6" << std::endl;
    int nBlocks1 = n1/blockSize + (n1%blockSize == 0?0:1);
    int nBlocks2 = n2/blockSize + (n2%blockSize == 0?0:1);
    info << alignl << "nBlocks1 = " << nBlocks1 << std::endl;
    info << alignl << "nBlocks2 = " << nBlocks2 << std::endl;
    info << alignl << "blockSize = " << blockSize << std::endl;
    info << alignl << "n1 = " << n1 << std::endl;
    info << alignl << "n2 = " << n2 << std::endl;
    cuda_match6 <<< dim3(nBlocks1, nBlocks2), dim3(blockSize, blockSize) >>>
    		(profdb1_d, n1, profdb2_d, n2, back, part, results_d, lr_threshold);

    hipError_t cts = hipDeviceSynchronize();
    info << startl << "hipDeviceSynchronize() == " << cts << std::endl;
    hipError_t err = hipGetLastError();
    info << startl << "hipGetLastError() == " << err << std::endl;

    Assert2(cts == hipSuccess, "cuda_nm_match: M: hipDeviceSynchronize failed");

    t.stop();
    info << startl << "cuda_nm_match(): Kernel cuda_match6 took " << t << " seconds" << std::endl;

    // get result from device
    hipMemcpy(&results, results_d, sizeof(N2Results)*1, hipMemcpyDeviceToHost);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_match: N: hipDeviceSynchronize failed");

    info << startl << min(results.count, n2result_max) << " results copied from device" << std::endl;

	// clean up
	hipFree(results_d);
	hipFree(profdb1_data);
	hipFree(profdb2_data);
	hipFree(profdb1_d);
	hipFree(profdb2_d);
	hipFree(back_data);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_match: P: hipDeviceSynchronize failed");
}

// match REL(prof_db1) against prof_db2 (not necessarily the same size)
void cuda_nm_relmatch(
	std::vector<DProfile> &prof_db1,   // Profile database
	std::vector<DProfile> &prof_db2,   // Profile database
	CUDAMatchType   const &match_type, // Relative type
	DProfile               backh,      // Background as a half-matrices
    CudaLocusInfo   const &locus_info,
    DBackground            backv,      // Background as vectors
	N2Results &results,
	float lr_threshold,
	ArrayPart part)
{
	int n_floats_per_profile = backh.size;
	int n1 = prof_db1.size();
	int n2 = prof_db2.size();
	int n_floats1 = n_floats_per_profile * n1;
	int n_floats2 = n_floats_per_profile * n2;
	Assert(n_floats_per_profile == locus_info.profile_size);

//	Assert(part == full || n1 == n2);

	//
	// allocate DProfiles data and result on device
	//
    Timer t;

	// the result
	N2Results *results_d;
	Assert2(hipMalloc((void **) &results_d, sizeof(N2Results)*1) != hipErrorOutOfMemory,
			"cuda_nm_relmatch: AA: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_relmatch: A: hipDeviceSynchronize failed");
	Assert(results_d);

    t.stop();
    info << startl << "cuda_nm_relmatch(): hipMalloc at 1680 took " << t << " seconds" << std::endl;

	// the data in the profiles
	// NB we will start by using db2 as temporary storage for db1, so create it at size n1
	float *profdb1_data = 0;
	Assert2(hipMalloc((void **) &profdb1_data, sizeof(float)*n_floats1) != hipErrorOutOfMemory,
			"cuda_nm_relmatch: B: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_relmatch: C: hipDeviceSynchronize failed");
	Assert(profdb1_data);

	float *profdb2_data = 0;
	Assert2(hipMalloc((void **) &profdb2_data, sizeof(float)*n_floats1) != hipErrorOutOfMemory,
			"cuda_nm_relmatch: D: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_relmatch: E: hipDeviceSynchronize failed");
	Assert(profdb2_data);

	// the DProfiles themselves (containing pointers to the data)
	DProfile *profdb1_d = 0;
	Assert2(hipMalloc((void **) &profdb1_d, sizeof(DProfile)*n1) != hipErrorOutOfMemory,
			"cuda_nm_relmatch: F: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_relmatch: G: hipDeviceSynchronize failed");
	Assert(profdb1_d);

	DProfile *profdb2_d = 0;
	Assert2(hipMalloc((void **) &profdb2_d, sizeof(DProfile)*n1) != hipErrorOutOfMemory,
			"cuda_nm_relmatch: H: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_relmatch: I: hipDeviceSynchronize failed");
	Assert(profdb2_d);

	// the data for the background
	float *backh_data = 0;
	Assert2(hipMalloc((void **) &backh_data, sizeof(float)*n_floats_per_profile) != hipErrorOutOfMemory,
			"cuda_nm_relmatch: K: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_relmatch: L: hipDeviceSynchronize failed 6");
	Assert(backh_data);

	float *backv_data = 0;
	Assert2(hipMalloc((void **) &backv_data, sizeof(float)*locus_info.back_size) != hipErrorOutOfMemory,
			"cuda_nm_relmatch: M: hipMalloc failed");

	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_relmatch: N: hipDeviceSynchronize failed 7");
	Assert(backv_data);

    t.stop();
    info << startl << "cuda_nm_relmatch(): Allocating memory on device took " << t << " seconds" << std::endl;

	//
	// copy to device
	//
    t.start();

    // copy offsets
    COPY_OFFSETS();

	// zero the result
	results.count = 0;
	hipMemcpy(results_d, &results, sizeof(NResults)*1, hipMemcpyHostToDevice);

	// copy db1 data into db2 on the device
	for (int i=0; i<n1; ++i)
	{
		float *addr = profdb2_data + (i * n_floats_per_profile);

		hipMemcpy(addr, prof_db1[i].data, sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);

		// after copying, overwrite the data pointer with the device address
		prof_db1[i].data = addr;
	}

	// copy the DProfiles themselves (containing pointers to the data) for db2
	hipMemcpy(profdb2_d, &(prof_db1[0]), sizeof(DProfile)*n1, hipMemcpyHostToDevice);

	// create array of DProfiles for db1 on the device
	for (int i=0; i<n1; ++i)
	{
		float *addr = profdb1_data + (i * n_floats_per_profile);

		// copy no data into db1 - this is where the relatives profiles will go

		// re-use the db1 data pointers
		prof_db1[i].data = addr;
	}

	// copy the DProfiles themselves (containing pointers to the data) for db1
	hipMemcpy(profdb1_d, &(prof_db1[0]), sizeof(DProfile)*n1, hipMemcpyHostToDevice);

	// the data for the background DProfile
	hipMemcpy(backh_data,   backh.data,  sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
	backh.data  = backh_data;

	// the data for the background DBackground
	hipMemcpy(backv_data,   backv.data,  sizeof(float)*locus_info.back_size, hipMemcpyHostToDevice);
	backv.data  = backv_data;

	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_relmatch: L: hipDeviceSynchronize failed");

    t.stop();
    info << startl << "cuda_nm_relmatch(): Copying first dataset to device took " << t << " seconds" << std::endl;

	// call kernel to calculate relative profile of first db. (This now goes in the db1 position on the device)
    t.start();
    if (match_type.m_rel_type == sibling_t)
    {
    	info << startl << "launching kernel cuda_sib" << std::endl;
    }
    else
    {
    	info << startl << "launching kernel cuda_rel" << std::endl;
    }

    int nBlocks = n1/blockSize + (n1%blockSize == 0?0:1);
    info << alignl << "nBlocks = " << nBlocks << std::endl;
    info << alignl << "blockSize = " << blockSize << std::endl;
    info << alignl << "n1 = " << n1 << std::endl;

    Assert2(match_type.m_rel_type != ident_t, "cuda_nm_relmatch: P2: called with m_rel_type == ident_t");

    if (match_type.m_rel_type == sibling_t)
    {
		cuda_sib <<< dim3(nBlocks), dim3(blockSize) >>>
				(profdb2_d, profdb1_d, n1, backv);
    }
    else if (match_type.m_rel_type == gen_t)
    {
		cuda_genrc <<< dim3(nBlocks), dim3(blockSize) >>>
				(profdb2_d, profdb1_d, n1, match_type.m_a1, match_type.m_b1, match_type.m_a2, match_type.m_b2, backv);
    }
    else if (match_type.m_rel_type == inv_t)
    {
    	bool inverse = true;
		cuda_genrc <<< dim3(nBlocks), dim3(blockSize) >>>
				(profdb2_d, profdb1_d, n1, match_type.m_a1, match_type.m_b1, match_type.m_a2, match_type.m_b2, backv, inverse);
    }
    else
    {
		cuda_rel <<< dim3(nBlocks), dim3(blockSize) >>>
				(profdb2_d, profdb1_d, n1, match_type.m_path1steps, match_type.m_path2steps, backv);
    }

    hipError_t cts = hipDeviceSynchronize();
    info << startl << "hipDeviceSynchronize() == " << cts << std::endl;
    hipError_t err = hipGetLastError();
    info << startl << "hipGetLastError() == " << err << std::endl;

    Assert2(cts == hipSuccess, "cuda_nm_relmatch: Q: hipDeviceSynchronize failed 9");

    t.stop();
    info << startl << "Kernel " << ((match_type.m_rel_type == sibling_t) ? "cuda_sib" : "cuda_rel") << " took " << t << " seconds" << std::endl;

    // deallocate db2 on the device and reallocate it the right size
    t.start();

	hipFree(profdb2_data);
	hipFree(profdb2_d);

    profdb2_data = 0;
	Assert2(hipMalloc((void **) &profdb2_data, sizeof(float)*n_floats2) != hipErrorOutOfMemory,
			"cuda_nm_relmatch: D: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_relmatch: E2: hipDeviceSynchronize failed");
	Assert(profdb2_data);

	profdb2_d = 0;
	Assert2(hipMalloc((void **) &profdb2_d, sizeof(DProfile)*n2) != hipErrorOutOfMemory,
			"cuda_nm_relmatch: H: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_relmatch: I2: hipDeviceSynchronize failed");
	Assert(profdb2_d);

    t.stop();
    info << startl << "cuda_nm_relmatch(): Allocating memory on device took " << t << " seconds" << std::endl;

    // copy second database to db2 on the device
	t.start();

	for (int i=0; i<n2; ++i)
	{
		float *device_addr = profdb2_data + (i * n_floats_per_profile);
		hipMemcpy(device_addr, prof_db2[i].data, sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
		prof_db2[i].data = device_addr;
	}

	// the DProfiles themselves (containing pointers to the data)
	hipMemcpy(profdb2_d, &(prof_db2[0]), sizeof(DProfile)*n2, hipMemcpyHostToDevice);

    t.stop();
    info << startl << "cuda_nm_relmatch(): Copying second dataset to device took " << t << " seconds" << std::endl;

    // call solution kernel
    t.start();

    info << startl << "launching kernel cuda_match6" << std::endl;
    int nBlocks1 = n1/blockSize + (n1%blockSize == 0?0:1);
    int nBlocks2 = n2/blockSize + (n2%blockSize == 0?0:1);
    info << alignl << "nBlocks1 = " << nBlocks1 << std::endl;
    info << alignl << "nBlocks2 = " << nBlocks2 << std::endl;
    info << alignl << "blockSize = " << blockSize << std::endl;
    info << alignl << "n1 = " << n1 << std::endl;
    info << alignl << "n2 = " << n2 << std::endl;
    cuda_match6 <<< dim3(nBlocks1, nBlocks2), dim3(blockSize, blockSize) >>>
    		(profdb1_d, n1, profdb2_d, n2, backh, part, results_d, lr_threshold);

    cts = hipDeviceSynchronize();
    info << startl << "hipDeviceSynchronize() == " << cts << std::endl;
    err = hipGetLastError();
    info << startl << "hipGetLastError() == " << err << std::endl;

    Assert2(cts == hipSuccess, "cuda_nm_relmatch: M: hipDeviceSynchronize failed");

    t.stop();
    info << startl << "cuda_nm_relmatch(): Kernel cuda_match6 took " << t << " seconds" << std::endl;

    // get result from device
    hipMemcpy(&results, results_d, sizeof(N2Results)*1, hipMemcpyDeviceToHost);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_relmatch: N: hipDeviceSynchronize failed");

    info << startl << min(results.count, n2result_max) << " results copied from device" << std::endl;

    // clean up
	hipFree(results_d);
	hipFree(profdb1_data);
	hipFree(profdb2_data);
	hipFree(profdb1_d);
	hipFree(profdb2_d);
	hipFree(backh_data);
	hipFree(backv_data);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_nm_relmatch: P: hipDeviceSynchronize failed");
}

// load data and allocate equal space for the relative data
// call kernel to fill relative data
// call match kernel, upper mode, point to the separate arrays
void cuda_n2relmatch(
	std::vector<DProfile> &prof_db,    // Profile database
	CUDAMatchType   const &match_type, // Relative type
	DProfile               backh,      // Background as a half-matrices
    CudaLocusInfo   const &locus_info,
    DBackground            backv,      // Background as vectors
	N2Results             &results,
	float                  lr_threshold)
{
	int n_floats_per_profile = backh.size;
	int n_profiles = prof_db.size();
	int n_floats = n_floats_per_profile * n_profiles;
	Assert(n_floats_per_profile == locus_info.profile_size);
	Assert(backv.size == locus_info.back_size);
	//
	// allocate DProfiles data and result on device
	//
    Timer t;

	// the result
	N2Results *results_d;
	Assert2(hipMalloc((void **) &results_d, sizeof(N2Results)*1) != hipErrorOutOfMemory,
			"cuda_n2relmatch: A: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2relmatch: B: hipDeviceSynchronize failed 1");
	Assert(results_d);

	// the data in the profiles (two copies: originals and relatives)
	float *profdb1_data = 0;
	Assert2(hipMalloc((void **) &profdb1_data, sizeof(float)*n_floats) != hipErrorOutOfMemory,
			"cuda_n2relmatch: C: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2relmatch: D: hipDeviceSynchronize failed 2");
	Assert(profdb1_data);

	float *profdb2_data = 0;
	Assert2(hipMalloc((void **) &profdb2_data, sizeof(float)*n_floats) != hipErrorOutOfMemory,
			"cuda_n2relmatch: E: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2relmatch: F: hipDeviceSynchronize failed 3");
	Assert(profdb2_data);

	// the DProfiles themselves, containing pointers to the data (two, ditto)
	DProfile *profdb1_d = 0;
	Assert2(hipMalloc((void **) &profdb1_d, sizeof(DProfile)*n_profiles) != hipErrorOutOfMemory,
			"cuda_n2relmatch: G: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2relmatch: H: hipDeviceSynchronize failed 4");
	Assert(profdb1_d);

	DProfile *profdb2_d = 0;
	Assert2(hipMalloc((void **) &profdb2_d, sizeof(DProfile)*n_profiles) != hipErrorOutOfMemory,
			"cuda_n2relmatch: I: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2relmatch: J: hipDeviceSynchronize failed 5");
	Assert(profdb2_d);

	// the data for the background
	float *backh_data = 0;
	Assert2(hipMalloc((void **) &backh_data, sizeof(float)*n_floats_per_profile) != hipErrorOutOfMemory,
			"cuda_n2relmatch: K: hipMalloc failed");
	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2relmatch: L: hipDeviceSynchronize failed 6");
	Assert(backh_data);

	float *backv_data = 0;
	Assert2(hipMalloc((void **) &backv_data, sizeof(float)*locus_info.back_size) != hipErrorOutOfMemory,
			"cuda_n2relmatch: M: hipMalloc failed");

	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2relmatch: N: hipDeviceSynchronize failed 7");
	Assert(backv_data);

    t.stop();
    info << startl << "cuda_n2relmatch(): Allocating memory on device took " << t << " seconds" << std::endl;

	//
	// copy to device (copy original, allocate space for relative)
	//
    t.start();

    // copy offsets
    COPY_OFFSETS();

	// zero the result
	results.count = 0;
	hipMemcpy(results_d, &results, sizeof(NResults)*1, hipMemcpyHostToDevice);

	// Data for each DProfile. After copying each data array, copy the device address into the DProfile.
	for (int i=0; i<n_profiles; ++i)
	{
		float *addr = profdb1_data + (i * n_floats_per_profile);
		hipMemcpy(addr, prof_db[i].data, sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
		prof_db[i].data = addr;
	}

	// the DProfiles themselves (containing pointers to the data)
	hipMemcpy(profdb1_d, &(prof_db[0]), sizeof(DProfile)*n_profiles, hipMemcpyHostToDevice);

	// construct and copy over addresses for the sibling database
	for (int i=0; i<n_profiles; ++i)
	{
		float *addr = profdb2_data + (i * n_floats_per_profile);
		// no data to copy: profdb2 will be uninitialized on the device
		prof_db[i].data = addr;
	}
	hipMemcpy(profdb2_d, &(prof_db[0]), sizeof(DProfile)*n_profiles, hipMemcpyHostToDevice);

	// the data for the background DProfile
	hipMemcpy(backh_data,   backh.data,  sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
	backh.data  = backh_data;

	// the data for the background DBackground
	hipMemcpy(backv_data,   backv.data,  sizeof(float)*backv.size, hipMemcpyHostToDevice);
	backv.data  = backv_data;

	Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2relmatch: P: hipDeviceSynchronize failed 8");

    t.stop();
    info << startl << "cuda_n2relmatch(): Copying data to device took " << t << " seconds" << std::endl;

	// call kernel to calculate relative profile
    t.start();

    info << startl << "launching kernel cuda_sib" << std::endl;
    int nBlocks = n_profiles/blockSize + (n_profiles%blockSize == 0?0:1);
    info << alignl << "nBlocks = " << nBlocks << std::endl;
    info << alignl << "blockSize = " << blockSize << std::endl;
    info << alignl << "n_profiles = " << n_profiles << std::endl;

    Assert2(match_type.m_rel_type != ident_t, "cuda_n2relmatch: P2: called with m_rel_type == ident_t");

    if (match_type.m_rel_type == sibling_t)
    {
		cuda_sib <<< dim3(nBlocks), dim3(blockSize) >>>
				(profdb1_d, profdb2_d, n_profiles, backv);
    }
    else if (match_type.m_rel_type == gen_t)
    {
		cuda_genrc <<< dim3(nBlocks), dim3(blockSize) >>>
				(profdb1_d, profdb2_d, n_profiles, match_type.m_a1, match_type.m_b1, match_type.m_a2, match_type.m_b2, backv);
    }
    else if (match_type.m_rel_type == inv_t)
    {
    	bool inverse = true;
		cuda_genrc <<< dim3(nBlocks), dim3(blockSize) >>>
				(profdb1_d, profdb2_d, n_profiles, match_type.m_a1, match_type.m_b1, match_type.m_a2, match_type.m_b2, backv, inverse);

    }
    else
    {
		cuda_rel <<< dim3(nBlocks), dim3(blockSize) >>>
				(profdb1_d, profdb2_d, n_profiles, match_type.m_path1steps, match_type.m_path2steps, backv);
    }

    hipError_t cts = hipDeviceSynchronize();
    info << startl << "hipDeviceSynchronize() == " << cts << std::endl;
    hipError_t err = hipGetLastError();
    info << startl << "hipGetLastError() == " << err << std::endl;

    Assert2(cts == hipSuccess, "cuda_n2relmatch: Q: hipDeviceSynchronize failed 9");

    t.stop();
    info << startl << "Kernel " << ((match_type.m_rel_type == sibling_t) ? "cuda_sib" : "cuda_rel") << " took " << t << " seconds" << std::endl;

	// call match kernel. upper mode, point to original and sibling arrays
    t.start();
    info << startl << "launching kernel cuda_match6" << std::endl;
    nBlocks = n_profiles/blockSize + (n_profiles%blockSize == 0?0:1);
    info << alignl << "nBlocks = " << nBlocks << std::endl;
    info << alignl << "blockSize = " << blockSize << std::endl;
    info << alignl << "n_profiles = " << n_profiles << std::endl;
    cuda_match6 <<< dim3(nBlocks, nBlocks), dim3(blockSize, blockSize) >>>
    		(profdb1_d, n_profiles, profdb2_d, n_profiles, backh, upper, results_d, lr_threshold);

    cts = hipDeviceSynchronize();
    info << startl << "hipDeviceSynchronize() == " << cts << std::endl;
    err = hipGetLastError();
    info << startl << "hipGetLastError() == " << err << std::endl;

    Assert2(cts == hipSuccess, "cuda_n2relmatch: R: hipDeviceSynchronize failed 10");

    t.stop();
    info << startl << "cuda_n2relmatch(): Kernel cuda_match6 took " << t << " seconds" << std::endl;

    // get result from device
    hipMemcpy(&results, results_d, sizeof(N2Results)*1, hipMemcpyDeviceToHost);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2relmatch: S: hipDeviceSynchronize failed 11");

    info << startl << min(results.count, n2result_max) << " results copied from device" << std::endl;

	// clean up
	hipFree(results_d);
	hipFree(profdb1_data);
	hipFree(profdb2_data);
	hipFree(profdb1_d);
	hipFree(profdb2_d);
	hipFree(backh_data);
	hipFree(backv_data);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_n2relmatch: T: hipDeviceSynchronize failed 12");
}

#endif

// this version calls a kernel to do an n_profile/n_profile match
void cuda_lr(std::vector<DProfile> &prof_db, // Profile database
	         DProfile back,                  // Background
		     CudaLocusInfo   const &locus_info,
	         N2Results &results,
	         float lr_threshold)
{
	int n_floats_per_profile = back.size;
	int n_profiles = prof_db.size();
	int n_floats = n_floats_per_profile * n_profiles;
	Assert(n_floats_per_profile == locus_info.profile_size);

	//
    // allocate DProfiles data and result on device
	//

	// the result
	N2Results *results_d;
    Assert2(hipMalloc((void **) &results_d, sizeof(N2Results)*1) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(results_d);

    // the data in the profiles
	float *profdb_data = 0;
    Assert2(hipMalloc((void **) &profdb_data, sizeof(float)*n_floats) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(profdb_data);

    // the DProfiles themselves (containing pointers to the data)
    DProfile *profdb_d = 0;
    Assert2(hipMalloc((void **) &profdb_d, sizeof(DProfile)*n_profiles) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(profdb_d);

    // the data for the background
	float *back_data = 0;
    Assert2(hipMalloc((void **) &back_data, sizeof(float)*n_floats_per_profile) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(back_data);

    //
    // copy to device
    //

    // copy offsets
    COPY_OFFSETS();

    // zero the result
    results.count = 0;
    hipMemcpy(results_d, &results, sizeof(NResults)*1, hipMemcpyHostToDevice);

	// Data for each DProfile. After copying each data array, copy the device address into the DProfile.
    for (int i=0; i<n_profiles; ++i)
    {
    	float *addr = profdb_data + (i * n_floats_per_profile);
    	hipMemcpy(addr, prof_db[i].data, sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
    	prof_db[i].data = addr;
    }

    // the DProfiles themselves (containing pointers to the data)
    hipMemcpy(profdb_d, &(prof_db[0]), sizeof(DProfile)*n_profiles, hipMemcpyHostToDevice);

    // the data for the background
    hipMemcpy(back_data,   back.data,  sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
    back.data  = back_data;

    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: A: hipDeviceSynchronize failed");

	// compute LR
    Timer t;
#if 0
    // each thread does N matches
    int nBlocks = n_profiles/blockSize + (n_profiles%blockSize == 0?0:1);
    info << startl << "launching kernel cuda_match4" << std::endl;
    info << alignl << "nBlocks = " << nBlocks << std::endl;
    info << alignl << "blockSize = " << blockSize << std::endl;
    info << alignl << "n_profiles = " << n_profiles << std::endl;
    cuda_match4 <<< dim3(nBlocks), dim3(blockSize) >>> (profdb_d, n_profiles, back, results_d, lr_threshold);
#else
    // each thread does 1 match
#ifdef CUDA_MATCH5_HALF
    info << startl << "launching kernel cuda_match5 (Half addressing)" << std::endl;
    int nxBlocks = n_profiles/(2*blockSize) + ((n_profiles/2)%blockSize == 0?0:1);
    int nyBlocks = (n_profiles-1)/blockSize + ((n_profiles-1)%blockSize == 0?0:1);
    info << alignl << "nxBlocks = " << nxBlocks << std::endl;
    info << alignl << "nyBlocks = " << nyBlocks << std::endl;
    info << alignl << "blockSize = " << blockSize << std::endl;
    info << alignl << "n_profiles = " << n_profiles << std::endl;
    cuda_match5 <<< dim3(nxBlocks, nyBlocks), dim3(blockSize, blockSize) >>> (profdb_d, n_profiles, back, results_d, lr_threshold);

#else
    info << startl << "launching kernel cuda_match5 (Full addressing)" << std::endl;
    int nBlocks = n_profiles/blockSize + (n_profiles%blockSize == 0?0:1);
    info << alignl << "nBlocks = " << nBlocks << std::endl;
    info << alignl << "blockSize = " << blockSize << std::endl;
    info << alignl << "n_profiles = " << n_profiles << std::endl;
    cuda_match5 <<< dim3(nBlocks, nBlocks), dim3(blockSize, blockSize) >>> (profdb_d, n_profiles, back, results_d, lr_threshold);

#endif
#endif
    hipError_t cts = hipDeviceSynchronize();
    info << startl << "hipDeviceSynchronize() == " << cts << std::endl;
    hipError_t err = hipGetLastError();
    info << startl << "hipGetLastError() == " << err << std::endl;

    Assert2(cts == hipSuccess, "cuda_lr: B: hipDeviceSynchronize failed");

    t.stop();
    info << startl << "Kernel took " << t << " seconds" << std::endl;

    // get result from device
    hipMemcpy(&results, results_d, sizeof(N2Results)*1, hipMemcpyDeviceToHost);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: C: hipDeviceSynchronize failed");

    info << startl << min(results.count, n2result_max) << " results copied from device" << std::endl;

	// clean up
	hipFree(results_d);
	hipFree(profdb_data);
	hipFree(profdb_d);
	hipFree(back_data);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: D: hipDeviceSynchronize failed");
}

// calls a kernel to do an n_profile/profile match
void cuda_n_match(
	std::vector<DProfile> &prof_db, // Profile database
	DProfile prof,                  // Single profile to compare it with
	DProfile back,                  // Background
    CudaLocusInfo const &locus_info,
	NResults &results,
	float lr_threshold)
{
	int n_floats_per_profile = back.size;
	int n_profiles = prof_db.size();
	int n_floats = n_floats_per_profile * n_profiles;
	Assert(n_floats_per_profile == locus_info.profile_size);
	Assert(prof.size == locus_info.profile_size);

	//
    // allocate DProfiles data and result on device
	//
    Timer t;

	// the result
	NResults *results_d;
    Assert2(hipMalloc((void **) &results_d, sizeof(NResults)*1) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(results_d);

    // the data in the profiles
	float *profdb_data = 0;
    Assert2(hipMalloc((void **) &profdb_data, sizeof(float)*n_floats) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(profdb_data);

    // the DProfiles themselves (containing pointers to the data)
    DProfile *profdb_d = 0;
    Assert2(hipMalloc((void **) &profdb_d, sizeof(DProfile)*n_profiles) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(profdb_d);

    // the data for the test profile
    float *prof_data = 0;
    Assert2(hipMalloc((void **) &prof_data, sizeof(float)*n_floats_per_profile) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(prof_data);

    // the data for the background
	float *back_data = 0;
    Assert2(hipMalloc((void **) &back_data, sizeof(float)*n_floats_per_profile) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(back_data);

    t.stop();
    info << startl << "cuda_n_match(): Allocating memory on device took " << t << " seconds" << std::endl;

    //
    // copy to device
    //
    t.start();

    // copy offsets
    COPY_OFFSETS();

	// zero the result
    results.count = 0;
    hipMemcpy(results_d, &results, sizeof(NResults)*1, hipMemcpyHostToDevice);

	// Data for each DProfile. After copying each data array, copy the device address into the DProfile.
    for (int i=0; i<n_profiles; ++i)
    {
    	float *addr = profdb_data + (i * n_floats_per_profile);
    	hipMemcpy(addr, prof_db[i].data, sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
    	prof_db[i].data = addr;
    }

    // the DProfiles themselves (containing pointers to the data)
    hipMemcpy(profdb_d, &(prof_db[0]), sizeof(DProfile)*n_profiles, hipMemcpyHostToDevice);

    // the data for the test profile
    hipMemcpy(prof_data, prof.data, sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
    prof.data = prof_data;

    // the data for the background
    hipMemcpy(back_data,   back.data,  sizeof(float)*n_floats_per_profile, hipMemcpyHostToDevice);
    back.data  = back_data;

    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: A: hipDeviceSynchronize failed");

    t.stop();
    info << startl << "cuda_n_match(): Copying data to device took " << t << " seconds" << std::endl;

	// Call kernel to compute LRs
    t.start();

    int nBlocks = n_profiles/blockSize + (n_profiles%blockSize == 0?0:1);

    info << startl << "launching kernel cuda_match3" << std::endl;
    info << alignl << "nBlocks = " << nBlocks << std::endl;
    info << alignl << "blockSize = " << blockSize << std::endl;
    info << alignl << "n_profiles = " << n_profiles << std::endl;
    cuda_match3 <<< dim3(nBlocks), dim3(blockSize) >>> (profdb_d, n_profiles, prof, back, results_d, lr_threshold);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: B: hipDeviceSynchronize failed");

    t.stop();
    info << startl << "cuda_n_match(): Kernel cuda_match3 took " << t << " seconds" << std::endl;

    // get result from device
    hipMemcpy(&results, results_d, sizeof(NResults)*1, hipMemcpyDeviceToHost);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: C: hipDeviceSynchronize failed");

    info << startl << min(results.count, n2result_max) << " results copied from device" << std::endl;

	// clean up
	hipFree(results_d);
	hipFree(profdb_data);
	hipFree(profdb_d);
	hipFree(prof_data);
	hipFree(back_data);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: D: hipDeviceSynchronize failed");
}

// this version calls a kernel to do a single profile/profile match
double cuda_lr(
		ConstDProfile prof1,
		ConstDProfile prof2,
		ConstDProfile back,
	    CudaLocusInfo const &locus_info)
{
	float result_h = 0;
	int n = back.size;
	Assert(n == locus_info.profile_size);
	Assert(prof1.size == locus_info.profile_size);
	Assert(prof2.size == locus_info.profile_size);

    // allocate DProfiles data and result on device
    float *result_d = 0;     // pointer to device memory
    Assert2(hipMalloc((void **) &result_d, sizeof(float)*1) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(result_d);

	float *prof1_d = 0;
    Assert2(hipMalloc((void **) &prof1_d, sizeof(float)*n) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(prof1_d);

	float *prof2_d = 0;
    Assert2(hipMalloc((void **) &prof2_d, sizeof(float)*n) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(prof2_d);

	float *back_d = 0;
    Assert2(hipMalloc((void **) &back_d, sizeof(float)*n) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(back_d);

    // copy offsets
    COPY_OFFSETS();

	// load onto device
    hipMemcpy(result_d, &result_h, sizeof(float)*1, hipMemcpyHostToDevice);
    hipMemcpy(prof1_d, prof1.data, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(prof2_d, prof2.data, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(back_d,  back.data,  sizeof(float)*n, hipMemcpyHostToDevice);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");

    // store the device addresses
    DProfile p1, p2, b;
    p1.data = prof1_d; p1.size = locus_info.profile_size; // cuda_prof_size;
    p2.data = prof2_d; p2.size = locus_info.profile_size; // cuda_prof_size;
    b.data  = back_d;  b.size  = back.size;

	// compute LR
//    int blockSize = 8;
//    int nBlocks = n/blockSize + (n%blockSize == 0?0:1);
    int blockSize = 1;
    int nBlocks = 1;
    info << startl << "launching kernel cuda_match2" << std::endl;
    info << alignl << "nBlocks = " << nBlocks << std::endl;
    info << alignl << "blockSize = " << blockSize << std::endl;
    cuda_match2 <<< dim3(nBlocks), dim3(blockSize) >>> (p1, p2, b, result_d);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");

    // get result from device
    hipMemcpy(&result_h, result_d, sizeof(float)*1, hipMemcpyDeviceToHost);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");

    info << startl << 1 << " results copied from device" << std::endl;

	// clean up
	hipFree(result_d);
	hipFree(prof1_d);
	hipFree(prof2_d);
	hipFree(back_d);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");

    return (double)result_h;
}

// Performs a single locus/locus match (calculates LR) on the CUDA hardware.
// NB it is not efficient to do just one at a time - this is a test
//
// loc1, loc2 and background must have corresponding entries
double cuda_lr(const float *loc1, const float *loc2, const float *background, int n)
{
	float sum = 0;

    // allocate arrays and result on device
    float *loc1_d=0, *loc2_d=0, *back_d=0, *result_d=0;     // pointers to device memory
    Assert2(hipMalloc((void **) &loc1_d, sizeof(float)*n) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipMalloc((void **) &loc2_d, sizeof(float)*n) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipMalloc((void **) &back_d, sizeof(float)*n) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipMalloc((void **) &result_d, sizeof(float)*n) != hipErrorOutOfMemory,
    		"cuda_lr: hipMalloc failed");
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");
    Assert(loc1_d && loc2_d && back_d && result_d);

    // allocate result on host
    float *result_h = (float*)malloc(sizeof(float)*n);

	// load onto device
    hipMemcpy(loc1_d, loc1,      sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(loc2_d, loc2,      sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(back_d,  background, sizeof(float)*n, hipMemcpyHostToDevice);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");

	// compute LR
    int nBlocks = n/blockSize + (n%blockSize == 0?0:1);
    info << startl << "launching kernel cuda_match1" << std::endl;
    info << alignl << "nBlocks = " << nBlocks << std::endl;
    info << alignl << "blockSize = " << blockSize << std::endl;
    cuda_match1 <<< dim3(nBlocks), dim3(blockSize) >>> (loc1_d, loc2_d, back_d, n, result_d);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");

    // get result from device
    hipMemcpy(result_h, result_d, sizeof(float)*n, hipMemcpyDeviceToHost);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");

    info << startl << n << " results copied from device" << std::endl;

	// clean up
	hipFree(loc1_d);
	hipFree(loc2_d);
	hipFree(back_d);
	hipFree(result_d);
    Assert2(hipDeviceSynchronize() == hipSuccess, "cuda_lr: hipDeviceSynchronize failed");

    // sum on the CPU
    // to sum this on the device we need to do a reduction (see SDK reduction example)
	sum = 0;
	for (int i=0; i<n; ++i)
	{
		sum += result_h[i];
	}

	return sum;
}

__host__
void
runCudaSPMC(
	int 		nBlocks,
	int 		blockSize,
	DProfile 	*prof_db,    // profile dataset
	DProfile 	*spmc_db,    // correction factor matrices (to construct)
	int 		n,           // size of dataset
	DProfile    back,        // background matrix (4.4)
    DBackground backv,       // background vector (HW)
    float       theta)       // Fst
{
	cuda_spmc <<< dim3(nBlocks), dim3(blockSize) >>>
			(prof_db, spmc_db, n, back, backv, theta);
}

__host__
void
runCudaSib(
	int 		nBlocks,
	int 		blockSize,
	DProfile 	*prof_db,    // profile dataset
	DProfile 	*sib_db,     // sibling dataset (to construct)
	int 		n,           // size of dataset
	DBackground back,        // background
	const CudaSubPopModel &spm)
{
	cuda_sib <<< dim3(nBlocks), dim3(blockSize) >>>
			(prof_db, sib_db, n, back, spm);
}

__host__
void
runCudaGenrc(
	int 		nBlocks,
	int 		blockSize,
	DProfile 	*prof_db,    // profile dataset
	DProfile 	*rel_db,     // relative dataset (to construct)
	int 		n,           // size of dataset
	float 		a1,          // proportion of 'a' inherited on side 1
	float 		b1,          // proportion of 'b' inherited on side 1
	float 		a2,          // proportion of 'a' inherited on side 2
	float 		b2,          // proportion of 'b' inherited on side 2
	DBackground back,        // background
	bool inverse)
{
	cuda_genrc <<< dim3(nBlocks), dim3(blockSize) >>>
			(prof_db, rel_db, n, a1, b1, a2, b2, back, inverse);
}

__host__
void
runCudaRel(
	int 		nBlocks,
	int 		blockSize,
	DProfile 	*prof_db,    // profile dataset
	DProfile 	*rel_db,     // relative dataset (to construct)
	int 		n,           // size of dataset
	int 		d1,          // degree first path
	int 		d2,          // degree second path
	DBackground back,        // background
	const CudaSubPopModel &spm)
{
	cuda_rel <<< dim3(nBlocks), dim3(blockSize) >>>
			(prof_db, rel_db, n, d1, d2, back, spm);
}

